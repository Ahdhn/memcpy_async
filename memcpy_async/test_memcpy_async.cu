#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>
#include <cuda/barrier>

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <numeric>
#include <string>

#include "gtest/gtest.h"
#include "helper.h"


__global__ void without_memcpy_async(uint32_t*       global_out,
                                     uint32_t const* global_in,
                                     uint32_t        size,
                                     uint32_t        num_batch_per_block)
{
    auto grid  = cooperative_groups::this_grid();
    auto block = cooperative_groups::this_thread_block();

    // input size fits num_batch_per_block * grid_size
    assert(size == num_batch_per_block * grid.size());

    // block.size() * sizeof(int) bytes
    extern __shared__ uint32_t shared[];

    uint32_t local_idx = block.thread_rank();

    for (uint32_t batch = 0; batch < num_batch_per_block; ++batch) {
        // Compute the index of the current batch for this block in global
        // memory:
        uint32_t block_batch_idx =
            block.group_index().x * block.size() + grid.size() * batch;

        uint32_t global_idx = block_batch_idx + local_idx;

        shared[local_idx] = global_in[global_idx];

        // Wait for all copies to complete
        block.sync();

        // Compute and write result to global memory
        global_out[global_idx] =
            shared[local_idx] + shared[block.size() - local_idx - 1];

        // Wait for compute using shared memory to finish
        block.sync();
    }
}

__global__ void with_memcpy_async(uint32_t*       global_out,
                                  uint32_t const* global_in,
                                  uint32_t        size,
                                  uint32_t        num_batch_per_block)
{
    auto grid  = cooperative_groups::this_grid();
    auto block = cooperative_groups::this_thread_block();

    // input size fits num_batch_per_block * grid_size
    assert(size == num_batch_per_block * grid.size());

    // block.size() * sizeof(int) bytes
    extern __shared__ uint32_t shared[];

    uint32_t local_idx = block.thread_rank();

    for (uint32_t batch = 0; batch < num_batch_per_block; ++batch) {
        // Compute the index of the current batch for this block in global
        // memory:
        uint32_t block_batch_idx =
            block.group_index().x * block.size() + grid.size() * batch;

        uint32_t global_idx = block_batch_idx + local_idx;

        // Whole thread-group cooperatively copies whole batch to shared memory:
        cooperative_groups::memcpy_async(block,
                                         shared,
                                         global_in + block_batch_idx,
                                         sizeof(uint32_t) * block.size());

        // Joins all threads, waits for all copies to complete
        cooperative_groups::wait(block);


        // Compute and write result to global memory
        global_out[global_idx] =
            shared[local_idx] + shared[block.size() - local_idx - 1];

        // Wait for compute using shared memory to finish
        block.sync();
    }
}


__global__ void with_memcpy_async_and_async_barrier(
    uint32_t*       global_out,
    uint32_t const* global_in,
    uint32_t        size,
    uint32_t        num_batch_per_block)
{
    using barrier = cuda::barrier<cuda::thread_scope::thread_scope_block>;
    __shared__ barrier bar;

    auto grid  = cooperative_groups::this_grid();
    auto block = cooperative_groups::this_thread_block();

    // input size fits num_batch_per_block * grid_size
    assert(size == num_batch_per_block * grid.size());

    // block.size() * sizeof(int) bytes
    extern __shared__ uint32_t shared[];

    uint32_t local_idx = block.thread_rank();

    if (block.thread_rank() == 0) {
        // Initialize the barrier with expected  arrival count
        init(&bar, block.size());
    }
    block.sync();

    for (uint32_t batch = 0; batch < num_batch_per_block; ++batch) {
        // Compute the index of the current batch for this block in global
        // memory:
        uint32_t block_batch_idx =
            block.group_index().x * block.size() + grid.size() * batch;

        uint32_t global_idx = block_batch_idx + local_idx;


        // Whole thread-group cooperatively copies whole batch to shared memory:
        cuda::memcpy_async(block,
                           shared,
                           global_in + block_batch_idx,
                           sizeof(uint32_t) * block.size(),
                           bar);

        // Waits for all copies to complete
        bar.arrive_and_wait();

        // Compute and write result to global memory
        global_out[global_idx] =
            shared[local_idx] + shared[block.size() - local_idx - 1];

        // Wait for compute using shared memory to finish
        block.sync();
    }
}

inline std::string method_to_string(uint32_t method)
{

    switch (method) {
        case 0:
            return "without_memcpy_async";
        case 1:
            return "with_memcpy_async";
        case 2:
            return "with_memcpy_async_and_async_barrier";

        default:
            return "unknown";
    }
}

TEST(Test, memcpy_async)
{
    uint32_t size                = 1024 * 1024;
    uint32_t block_size          = 256;
    uint32_t num_batch_per_block = 2;

    uint32_t *d_in(nullptr), *d_out(nullptr);
    CUDA_ERROR(hipMalloc((void**)&d_in, size * sizeof(uint32_t)));
    CUDA_ERROR(hipMalloc((void**)&d_out, size * sizeof(uint32_t)));

    std::vector<uint32_t> batch(block_size);
    std::iota(std::begin(batch), std::end(batch), 0);
    std::vector<uint32_t> h_in(size, 0), h_out(size, 0);
    uint32_t              num_segments = size / block_size;
    for (uint32_t i = 0; i < num_segments; ++i) {
        std::copy(batch.begin(), batch.end(), h_in.data() + i * batch.size());
    }
    uint32_t smem_bytes = block_size * sizeof(uint32_t);
    uint32_t grid_size  = DIVIDE_UP(size, block_size * num_batch_per_block);

    CUDA_ERROR(hipMemcpy(
        d_in, h_in.data(), size * sizeof(uint32_t), hipMemcpyHostToDevice));

    for (uint32_t method = 0; method < 3; ++method) {

        CUDA_ERROR(hipMemset(d_out, 0, size * sizeof(uint32_t)));

        CUDATimer timer;

        timer.start();
        if (method == 0) {
            without_memcpy_async<<<grid_size, block_size, smem_bytes>>>(
                d_out, d_in, size, num_batch_per_block);
        } else if (method == 1) {
            with_memcpy_async<<<grid_size, block_size, smem_bytes>>>(
                d_out, d_in, size, num_batch_per_block);
        } else if (method == 2) {
            with_memcpy_async_and_async_barrier<<<grid_size,
                                                  block_size,
                                                  smem_bytes>>>(
                d_out, d_in, size, num_batch_per_block);
        } else {
            std::cout << "Invalid method ID" << method << "\n";
            exit(EXIT_FAILURE);
        }
        timer.stop();
        CUDA_ERROR(hipMemcpy(h_out.data(),
                              d_out,
                              size * sizeof(uint32_t),
                              hipMemcpyDeviceToHost));

        for (uint32_t i = 0; i < h_out.size(); ++i) {
            ASSERT_EQ(h_out[i], block_size - 1);
        }


        hipFuncAttributes func_attr = hipFuncAttributes();
        if (method == 0) {
            CUDA_ERROR(hipFuncGetAttributes(&func_attr, reinterpret_cast<const void*>(without_memcpy_async)));
        } else if (method == 1) {
            CUDA_ERROR(hipFuncGetAttributes(&func_attr, reinterpret_cast<const void*>(with_memcpy_async)));
        } else if (method == 2) {
            CUDA_ERROR(hipFuncGetAttributes(
                &func_attr, reinterpret_cast<const void*>(with_memcpy_async_and_async_barrier)));
        }
        uint32_t num_reg_per_thread = static_cast<uint32_t>(func_attr.numRegs);
        uint32_t static_smem = static_cast<uint32_t>(func_attr.sharedSizeBytes);
        std::cout << method_to_string(method) << " took "
                  << timer.elapsed_millis() << " (ms), " << num_reg_per_thread
                  << " register/thread, and " << static_smem
                  << " (B) static shared memory" << std::endl;
    }

    auto err = hipDeviceSynchronize();
    EXPECT_EQ(err, hipSuccess);
}

int main(int argc, char** argv)
{
    ::testing::InitGoogleTest(&argc, argv);

    return RUN_ALL_TESTS();
}
